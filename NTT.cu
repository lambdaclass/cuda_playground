#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void mykernel(void){

}

void ntt_rec(int *x, int N, int w, int mod) {
	if (N <= 1) {
		return;
	}

	int odd[N/2];
	int even[N/2];
	for (int i = 0; i < N / 2; i++) {
		even[i] = x[i*2];
		odd[i] = x[i*2+1];
	}

	ntt_rec(even, N/2, (1LL * w * w) % mod, mod);
	ntt_rec(odd, N/2, (1LL * w * w) % mod, mod);

	// Calculate NTT
	int wn = 1;
	for (int k = 0; k < N / 2; k++) {
		int t = (1LL * wn * odd[k]) % mod;
		x[k] = (even[k] + t) % mod;
		x[N / 2 + k] = (even[k] - t + mod) % mod;
		wn = (1LL * wn * w) % mod;
	}
}

void ntt(int *x_in, int *x_out, int N) {
	const int mod = 998244353;
	const int w = 372528824;

	// Make copy of array and apply window
	for (int i = 0; i < N; i++) {
		x_out[i] = x_in[i];
		x_out[i] *= 1; // Window
	}

	ntt_rec(x_out, N, w, mod);
}

/**
 * Host main routine
 */
extern  "C" {

  void main_ntt (int *x_in, int *x_out, int n) {
      ntt(x_in, x_out, n);
			printf("Memory pos xout %d\n", x_out);
			for(int i = 0; i < n; i++){
				printf("%d\n", x_out[i]);
			}
  }

}
