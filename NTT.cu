#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"

/* ---------------------------------------------------------------------------
** Basic implementation of Cooley-Tukey FFT algorithm in C++
**
** Author: Darko Lukic <lukicdarkoo@gmail.com>
** -------------------------------------------------------------------------*/

#include <cmath>

__global__ void mykernel(void){

}

void ntt_rec(int *x, int N, int w, int mod) {
	if (N <= 1) {
		return;
	}

	int odd[N/2];
	int even[N/2];
	for (int i = 0; i < N / 2; i++) {
		even[i] = x[i*2];
		odd[i] = x[i*2+1];
	}

	ntt_rec(even, N/2, (1LL * w * w) % mod, mod);
	ntt_rec(odd, N/2, (1LL * w * w) % mod, mod);

	// Calculate NTT
	int wn = 1;
	for (int k = 0; k < N / 2; k++) {
		int t = (1LL * wn * odd[k]) % mod;
		x[k] = (even[k] + t) % mod;
		x[N / 2 + k] = (even[k] - t + mod) % mod;
		wn = (1LL * wn * w) % mod;
	}
}

void ntt(int *x_in, int *x_out, int N) {
	const int mod = 998244353;
	const int w = 3;

	// Make copy of array and apply window
	for (int i = 0; i < N; i++) {
		x_out[i] = x_in[i];
		x_out[i] *= 1; // Window
	}

	ntt_rec(x_out, N, w, mod);
}

/**
 * Host main routine
 */
extern  "C" {

  void main_ntt (int *x_in, int n) {
      printf("Malloc of %d elements\n", n);
      int *x_out = (int*)malloc(sizeof(int) * n);

      for(int i = 0; i < n; i++){
        printf("%d\n", x_in[i]);
      }

      printf("Calling function\n");
      ntt(x_in, x_out, n);

      for(int i = 0; i < n; i++){
        int elem = x_out[i];
        printf("%d\n", elem);
      }

      printf("Free\n");
      free(x_out);
  }

}