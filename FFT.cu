#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"

/* ---------------------------------------------------------------------------
** Basic implementation of Cooley-Tukey FFT algorithm in C++
**
** Author: Darko Lukic <lukicdarkoo@gmail.com>
** -------------------------------------------------------------------------*/

#include <cmath>
#include <complex>

__global__ void mykernel(void){

}

void fft_rec(std::complex<double> *x, int N) {
	// Check if it is splitted enough
	if (N <= 1) {
		return;
	}

	// Split even and odd
	std::complex<double> odd[N/2];
	std::complex<double> even[N/2];
	for (int i = 0; i < N / 2; i++) {
		even[i] = x[i*2];
		odd[i] = x[i*2+1];
	}

	// Split on tasks
	fft_rec(even, N/2);
	fft_rec(odd, N/2);


	// Calculate DFT
	for (int k = 0; k < N / 2; k++) {
		std::complex<double> t = exp(std::complex<double>(0, -2 * M_PI * k / N)) * odd[k];
		x[k] = even[k] + t;
		x[N / 2 + k] = even[k] - t;
	}
}

void fft(int *x_in,
	std::complex<double> *x_out,
	int N) {

	// Make copy of array and apply window
	for (int i = 0; i < N; i++) {
		x_out[i] = std::complex<double>(x_in[i], 0);
		x_out[i] *= 1; // Window
	}

	// Start recursion
	fft_rec(x_out, N);
}

/**
 * Host main routine
 */
extern  "C" {

  int main_fft (int elements) {
      srand(time(NULL));

      printf("Malloc of %d elements\n", elements);
      int *x_in = (int*)malloc(sizeof(int) * elements);
      std::complex<double> *x_out = (std::complex<double>*)malloc(sizeof(std::complex<double>) * elements);

      printf("Set random elements\n");
      for(int i = 0; i < elements; i++){
        x_in[i] = (rand() % 10);
      }

      for(int i = 0; i < elements; i++){
        printf("%d\n", x_in[i]);
      }

      printf("Calling function\n");
      fft(x_in, x_out, elements);

      for(int i = 0; i < elements; i++){
        std::complex<double> elem = x_out[i];
        printf("%f%+fi\n", std::real(elem), std::imag(elem));
      }

      printf("Free\n");
      free(x_in);
      free(x_out);

      printf("Done\n");
      return 0;
  }

}
