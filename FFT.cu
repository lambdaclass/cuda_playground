#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"

/* ---------------------------------------------------------------------------
** Basic implementation of Cooley-Tukey FFT algorithm in C++
**
** Author: Darko Lukic <lukicdarkoo@gmail.com>
** -------------------------------------------------------------------------*/

#include <cmath>
#include <complex>

__global__ void mykernel(void){

}

void fft(int *x_in,
	std::complex<double> *x_out,
	int N) {

	// Make copy of array and apply window
	for (int i = 0; i < N; i++) {
		x_out[i] = std::complex<double>(x_in[i], 0);
		x_out[i] *= 1; // Window
	}

	// Start recursion
	fft_rec(x_out, N);
}

void fft_rec(std::complex<double> *x, int N) {
	// Check if it is splitted enough
	if (N <= 1) {
		return;
	}

	// Split even and odd
	std::complex<double> odd[N/2];
	std::complex<double> even[N/2];
	for (int i = 0; i < N / 2; i++) {
		even[i] = x[i*2];
		odd[i] = x[i*2+1];
	}

	// Split on tasks
	fft_rec(even, N/2);
	fft_rec(odd, N/2);


	// Calculate DFT
	for (int k = 0; k < N / 2; k++) {
		std::complex<double> t = exp(std::complex<double>(0, -2 * M_PI * k / N)) * odd[k];
		x[k] = even[k] + t;
		x[N / 2 + k] = even[k] - t;
	}
}

/**
 * Host main routine
 */
extern  "C" {

  int main_fft (void) {
      printf("Malloc of 1024 elements\n");
      int elements = 1024;
      int *x_in = malloc(sizeof(int) * elements);
      std::complex<double> *x_out = malloc(sizeof(std::complex<double>) * elements);
      printf("Calling function\n");
      printf("Free\n");
      free(x_in);
      free(x_out);
      printf("Done\n");
      return 0;
  }

}
