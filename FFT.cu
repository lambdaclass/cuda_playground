#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"

__global__ void mykernel(void){

}

void fft_rec(int *x, int N, int w, int mod) {
	if (N <= 1) {
		return;
	}

	int odd[N/2];
	int even[N/2];
	for (int i = 0; i < N / 2; i++) {
		even[i] = x[i*2];
		odd[i] = x[i*2+1];
	}

	fft_rec(even, N/2, (1LL * w * w) % mod, mod);
	fft_rec(odd, N/2, (1LL * w * w) % mod, mod);

	// Calculate FFT
	int wn = 1;
	for (int k = 0; k < N / 2; k++) {
		int t = (1LL * wn * odd[k]) % mod;
		x[k] = (even[k] + t) % mod;
		x[N / 2 + k] = (even[k] - t + mod) % mod;
		wn = (1LL * wn * w) % mod;
	}
}

void fft(int *x_in, int *x_out, int N) {
	const int mod = 998244353;
	const int w = 372528824;

	for (int i = 0; i < N; i++) {
		x_out[i] = x_in[i];
	}

	fft_rec(x_out, N, w, mod);
}

/**
 * Host main routine
 */
extern  "C" {

  void main_fft (int *x_in, int *x_out, int n) {
      fft(x_in, x_out, n);
  }

}
